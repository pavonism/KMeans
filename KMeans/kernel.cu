#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h> 
#include <sys/types.h> 
#include <sys/stat.h>
#include <fcntl.h>
#include <cassert>
#include <limits>
#include <chrono>
#include <random>

#define MAX_CHAR_PER_LINE 128
#define THREADS_PER_BLOCK 1024
#define MAX_ITERATIONS 500
#define DEBUG 1

hipError_t kMeansThrust(float* points, int clustersCount, int pointsCount, int dimNum, float threshold, float** clusters, int* iterations, int** memberships);
hipError_t kMeansReduce(float* points, int clustersCount, int pointsCount, int dimNum, float threshold, float** clusters, int* iterations, int** memberships);
float* readFile(
	char* filename,
	int* pointsCount,
	int* dimNum);
int writeFile(char* filename,
	int numClusters,
	int numObjs,
	int numCoords,
	float* clusters,
	int* membership);
void coalesceData(float** points, int pointsCount, int dimNum);
void unCoalesceData(float** points, int pointsCount, int dimNum);
void getMinMax(float* points, int pointsCount, int dimNum, float** maxCoordinates, float** minCoordinates);
void generateClusters(float* minCoordinates, float* maxCoordinates, float** clusters, int clusterCount, int dimNum);
void printClusters(float* data, int numObjs, int numCoords);
void generateRandomPoints(char* path, int pointsCount, int dimNum);
void generateRandomPointsNormal(char* path, int pointsCount, int dimNum, int groups);
template <class T>
hipError_t reduce(T* data, int n);
template <class T>
hipError_t reduceChunks(T* values, int n, hipStream_t* streams, int k);

__global__ void pointsDistance(float* points, float* clusters, int* membership, int* membershipDims, int* membershipChanged, int clustersCount, int pointsCount, int dimNum)
{
	int pointNum = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	float minDistance = FLT_MAX;
	int currentMembership = -1;

	if (pointNum >= pointsCount)
		return;

	for (size_t i = 0; i < clustersCount; i++)
	{
		float distance = 0;

		for (size_t j = 0; j < dimNum; j++)
		{
			float diff = clusters[j * clustersCount + i] - points[j * pointsCount + pointNum];
			distance += diff * diff;

		}

		if (distance < minDistance) {
			minDistance = distance;
			currentMembership = i;
		}
	}

	__syncthreads();
	membershipChanged[pointNum] = membership[pointNum] != currentMembership;

	if (membershipDims != NULL) {
		__syncthreads();
		for (size_t i = 0; i < dimNum; i++)
		{
			membershipDims[i * pointsCount + pointNum] = i * clustersCount + currentMembership;
		}
	}
	__syncthreads();
	membership[pointNum] = currentMembership;
}

__global__ void updateClusters(float* clusters, float* clusterSums, int* clusterMembersCount, int* clusterMembersCountKeys, int clusterCount, int dimNum, int usedClusters) {

	int threadId = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

	if (threadId >= usedClusters)
		return;

	int clusterId = clusterMembersCountKeys[threadId];
	int members = clusterMembersCount[threadId];

	for (size_t i = 0; i < dimNum; i++)
	{
		clusters[i * clusterCount + clusterId] = clusterSums[i * usedClusters + threadId] / members;
	}
}

template <unsigned int blockSize, class T>
__device__ void warpReduce(volatile T* sdata, unsigned int tid) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize, class T>
__global__ void reduceGlobal(T* g_idata, T* g_odata, unsigned int n) {
	extern __shared__ __align__(sizeof(T)) unsigned char memory[];
	T* sdata = reinterpret_cast<T*>(memory);
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;
	while (i < n) { sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSize; }
	__syncthreads();
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) warpReduce<blockSize, T>(sdata, tid);
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void pickCoordinate(float* points, float* clusters, int* membership, float* sums, int* membersCount, int pointsCount, int pointsCountPowerOfTwo, int clustersCount, int dimNum) {

	extern __shared__ float pointData[];
	int threadId = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	int member = -1;
	float value = 0;

	if (threadId > pointsCountPowerOfTwo)
		return;

	if (threadId < pointsCount) {
		member = membership[threadId];
	}

	for (size_t dimension = 0; dimension < dimNum; dimension++)
	{
		if (threadId < pointsCount) {
			value = points[dimension * pointsCount + threadId];
		}

		for (size_t cluster = 0; cluster < clustersCount; cluster++)
		{
			float currentValue = member == cluster ? value / membersCount[cluster] : 0;
			sums[((long long)cluster * dimNum + dimension) * pointsCount + threadId] = currentValue;
		}
	}
}

__global__ void pickMembership(int* membership, int* sums, int pointsCount, int pointsCountPowerOfTwo, int clustersCount) {

	extern __shared__ int data[];

	int threadId = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

	if (threadId > pointsCountPowerOfTwo)
		return;

	for (size_t k = 0; k < clustersCount; k++)
	{
		int member = threadId < pointsCount ? membership[threadId] : -1;
		member = member == k ? 1 : 0;
		sums[k * pointsCountPowerOfTwo + threadId] = member;
	}
}

template <class T>
__global__ void gather(T* data, int chunkSizes, int chunks) {

	if (threadIdx.x > chunks)
		return;

	data[threadIdx.x] = data[(long long)threadIdx.x * chunkSizes];;
}

int main(int argc, char** argv)
{
	// TODO: dwie wersje mogą się odpalać jedno po drugim
	if (argc < 5)
	{
		printf("Wrong number of arguments!\n");
		return EXIT_FAILURE;
	}

	char* fileName = argv[1];

	int clustersCount = atoi(argv[2]);
	int pointsCount = 0, dimNum;
	float threshold = atof(argv[3]);
	float* minCoordinates = NULL;
	float* maxCoordinates = NULL;
	float* clusters = NULL;
	int* memberships = NULL;
	hipError_t cudaStatus;

	printf("Reading file...\n");
	auto cpuStart = std::chrono::high_resolution_clock::now();
	float* points = readFile((char*)fileName, &pointsCount, &dimNum);
	auto cpuEnd = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff = cpuEnd - cpuStart;
	printf("Elapsed %fs...\n", diff.count());

	coalesceData(&points, pointsCount, dimNum);
	getMinMax(points, pointsCount, dimNum, &maxCoordinates, &minCoordinates);
	// TODO: ziarno dla generatora na inpucie
	generateClusters(minCoordinates, maxCoordinates, &clusters, clustersCount, dimNum);
	printClusters(clusters, clustersCount, dimNum);
	int iterations;

	if (strcmp(argv[4], "r") == 0) {
		cudaStatus = kMeansReduce(points, clustersCount, pointsCount, dimNum, threshold, &clusters, &iterations, &memberships);
	}
	else {
		cudaStatus = kMeansThrust(points, clustersCount, pointsCount, dimNum, threshold, &clusters, &iterations, &memberships);
	}

	if (cudaStatus == hipSuccess) {
		printf("Iterations: %d\n", iterations);
		unCoalesceData(&clusters, clustersCount, dimNum);
		writeFile(fileName, clustersCount, pointsCount, dimNum, clusters, memberships);
	}

	delete minCoordinates;
	delete maxCoordinates;
	delete clusters;
	delete memberships;
	return EXIT_SUCCESS;
}

hipError_t kMeansThrust(float* points, int clustersCount, int pointsCount, int dimNum, float threshold, float** clusters, int* iterations, int** memberships)
{
	float* dev_points = NULL;
	float* dev_points_sums = NULL;
	float* dev_clusters = NULL;
	float* dev_clustersSums = NULL;
	int* dev_membershipDims = NULL;
	int* dev_membership = NULL;
	int* dev_currentMembership = NULL;
	int* dev_membershipChanged = NULL;
	int* dev_clusterSizes = NULL;
	int* dev_clusterSizesKeys = NULL;
	float* dev_delta = NULL;
	float delta = FLT_MAX;
	hipError_t cudaStatus;

	int blocksCount = (int)ceil((float)pointsCount / THREADS_PER_BLOCK);
	int newClustersBlocksCount = (int)ceil((float)clustersCount / THREADS_PER_BLOCK);

	int nearestPowerOfTwo = 1;
	while (nearestPowerOfTwo < clustersCount)
		nearestPowerOfTwo *= 2;
	int iterationNumber = 0;

	printf("Preparing memory...\n");

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_points, pointsCount * dimNum * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_points_sums, pointsCount * dimNum * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_clusters, clustersCount * dimNum * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_clustersSums, clustersCount * dimNum * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_membershipDims, dimNum * pointsCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_membership, pointsCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_currentMembership, pointsCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_membershipChanged, pointsCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_delta, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_clusterSizes, clustersCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_clusterSizesKeys, clustersCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_points, points, pointsCount * dimNum * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_clusters, *clusters, clustersCount * dimNum * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemset(dev_membership, -1, pointsCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	printf("Calculating on GPU...\n");
	hipEventRecord(start, 0);
	do {
		printf(".");
		pointsDistance << <blocksCount, THREADS_PER_BLOCK >> > (dev_points, dev_clusters, dev_membership, dev_membershipDims, dev_membershipChanged, clustersCount, pointsCount, dimNum);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "pointsDistance launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_points_sums, dev_points, pointsCount * dimNum * sizeof(int), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointsDistance!\n", cudaStatus);
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_currentMembership, dev_membership, pointsCount * sizeof(int), hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		thrust::sort(thrust::device, dev_currentMembership, dev_currentMembership + pointsCount);
		auto pair = thrust::reduce_by_key(thrust::device, dev_currentMembership, dev_currentMembership + pointsCount, thrust::make_constant_iterator(1), dev_clusterSizesKeys, dev_clusterSizes);
		int usedClusters = pair.first - dev_clusterSizesKeys;
		thrust::sort_by_key(thrust::device, dev_membershipDims, dev_membershipDims + pointsCount * dimNum, dev_points_sums);
		thrust::reduce_by_key(thrust::device, dev_membershipDims, dev_membershipDims + pointsCount * dimNum, dev_points_sums, thrust::make_discard_iterator(), dev_clustersSums);

		updateClusters << <newClustersBlocksCount, THREADS_PER_BLOCK >> > (dev_clusters, dev_clustersSums, dev_clusterSizes, dev_clusterSizesKeys, clustersCount, dimNum, usedClusters);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "pointsDistance launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		delta = (float)thrust::reduce(thrust::device, dev_membershipChanged, dev_membershipChanged + pointsCount);
		delta /= pointsCount;

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointsDistance!\n", cudaStatus);
			goto Error;
		}

		iterationNumber++;
	} while (delta > threshold && iterationNumber < MAX_ITERATIONS);
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Elapsed %fs\n", time / 1000);

	*iterations = iterationNumber;

	cudaStatus = hipMemcpy(*clusters, dev_clusters, clustersCount * dimNum * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	*memberships = new int[pointsCount];

	cudaStatus = hipMemcpy(*memberships, dev_membership, pointsCount * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_points);
	hipFree(dev_points_sums);
	hipFree(dev_clusters);
	hipFree(dev_clustersSums);
	hipFree(dev_membershipDims);
	hipFree(dev_membership);
	hipFree(dev_currentMembership);
	hipFree(dev_membershipChanged);
	hipFree(dev_clusterSizes);
	hipFree(dev_clusterSizesKeys);
	hipFree(dev_delta);

	return cudaStatus;
}

hipError_t kMeansReduce(float* points, int clustersCount, int pointsCount, int dimNum, float threshold, float** clusters, int* iterations, int** memberships) {

	float* dev_points = NULL;
	float* dev_sums = NULL;
	float* dev_clusters = NULL;
	int* dev_membership = NULL;
	int* dev_membership_sums = NULL;
	int* dev_membershipChanged = NULL;
	int* dev_membership_result = NULL;
	hipStream_t* streams = new hipStream_t[clustersCount * dimNum];
	int* membersCount = new int[clustersCount];
	float delta = FLT_MAX;
	hipError_t cudaStatus;

	for (size_t k = 0; k < clustersCount * dimNum; k++)
	{
		hipStreamCreate(&streams[k]);
	}

	int blocksCount = (int)ceil((float)pointsCount / THREADS_PER_BLOCK);
	int pointsCountPowerOfTwo = 1;
	while (pointsCountPowerOfTwo < pointsCount) pointsCountPowerOfTwo *= 2;
	int reduceBlocksCount = (int)ceil((float)pointsCountPowerOfTwo / THREADS_PER_BLOCK);
	int iterationNumber = 0;

	printf("Preparing memory...\n");

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_points, pointsCount * dimNum * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_sums, clustersCount * dimNum * pointsCountPowerOfTwo * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_clusters, clustersCount * dimNum * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_membership, pointsCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_membership_result, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_membership_sums, clustersCount * pointsCountPowerOfTwo * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_membershipChanged, pointsCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_points, points, pointsCount * dimNum * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_clusters, *clusters, clustersCount * dimNum * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemset(dev_membership, -1, pointsCount * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	printf("Calculating on GPU...\n");
	hipEventRecord(start, 0);

	do
	{
		printf(".");
		pointsDistance << <blocksCount, THREADS_PER_BLOCK >> > (dev_points, dev_clusters, dev_membership, NULL, dev_membershipChanged, clustersCount, pointsCount, dimNum);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "pointsDistance launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointsDistance!\n", cudaStatus);
			goto Error;
		}

		delta = (float)thrust::reduce(thrust::device, dev_membershipChanged, dev_membershipChanged + pointsCount);
		delta /= pointsCount;

		if (delta < threshold || iterationNumber++ > MAX_ITERATIONS)
			break;
		//printf("Delta: %f", delta);

		pickMembership << <reduceBlocksCount, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(int) >> > (
			dev_membership,
			dev_membership_sums,
			pointsCount,
			pointsCountPowerOfTwo,
			clustersCount
			);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "countMembers launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching countMembers!\n", cudaStatus);
			goto Error;
		}

		cudaStatus = reduceChunks(dev_membership_sums, pointsCountPowerOfTwo, streams, clustersCount);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduce launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}


		pickCoordinate << <reduceBlocksCount, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(float) >> > (
			dev_points,
			dev_clusters,
			dev_membership,
			dev_sums,
			dev_membership_sums,
			pointsCount,
			pointsCountPowerOfTwo,
			clustersCount,
			dimNum
			);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "countMembers launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching countMembers!\n", cudaStatus);
			goto Error;
		}

		cudaStatus = reduceChunks(dev_sums, pointsCountPowerOfTwo, streams, clustersCount * dimNum);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduce launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		hipMemcpy(dev_clusters, dev_sums, clustersCount * dimNum * sizeof(float), hipMemcpyDeviceToDevice);

	} while (1);
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Elapsed %fs\n", time / 1000);

	*iterations = iterationNumber;

	cudaStatus = hipMemcpy(*clusters, dev_clusters, clustersCount * dimNum * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	*memberships = new int[pointsCount];

	cudaStatus = hipMemcpy(*memberships, dev_membership, pointsCount * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	hipFree(dev_points);
	hipFree(dev_clusters);
	hipFree(dev_membership);
	hipFree(dev_membership_sums);
	hipFree(dev_membership_result);
	hipFree(dev_clusters);
	hipFree(dev_membershipChanged);

	for (size_t k = 0; k < clustersCount; k++)
	{
		hipStreamDestroy(streams[k]);
	}
	delete streams;
	delete membersCount;

	return cudaStatus;
}

template <class T>
hipError_t reduce(T* data, int n) {

	int currentThreads = n;
	hipError_t cudaStatus;

	while (currentThreads > 1) {
		//printf("currentThreads: %d\n", currentThreads);
		int dimGrid = (int)ceil((float)currentThreads / THREADS_PER_BLOCK);
		int dimBlock = dimGrid == 1 ? currentThreads : THREADS_PER_BLOCK;
		int	smemSize = dimBlock * sizeof(T);

		switch (currentThreads / 2)
		{
		default:
			reduceGlobal<512> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 256:
			reduceGlobal<256> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 128:
			reduceGlobal<128> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 64:
			reduceGlobal< 64> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 32:
			reduceGlobal< 32> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 16:
			reduceGlobal< 16> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 8:
			reduceGlobal< 8> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 4:
			reduceGlobal< 4> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 2:
			reduceGlobal< 2> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		case 1:
			reduceGlobal< 1> << < dimGrid, dimBlock, smemSize >> > (data, data, currentThreads); break;
		}

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduceGlobal launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduceGlobal!\n", cudaStatus);
			return cudaStatus;
		}

		currentThreads /= THREADS_PER_BLOCK;
	}

	return hipSuccess;
}

template <class T>
hipError_t reduceChunks(T* values, int n, hipStream_t* streams, int k) {

	int currentThreads = n;
	hipError_t cudaStatus;

	while (currentThreads > 1) {
		//printf("currentThreads: %d\n", currentThreads);
		int dimGrid = (int)ceil((float)currentThreads / THREADS_PER_BLOCK);
		int dimBlock = dimGrid == 1 ? currentThreads : THREADS_PER_BLOCK;
		int	smemSize = dimBlock * sizeof(T);

		for (size_t i = 0; i < k; i++)
		{
			T* data = values + i * n;

			switch (currentThreads / 2)
			{
			default:
				reduceGlobal<512> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 256:
				reduceGlobal<256> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 128:
				reduceGlobal<128> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 64:
				reduceGlobal< 64> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 32:
				reduceGlobal< 32> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 16:
				reduceGlobal< 16> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 8:
				reduceGlobal< 8> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 4:
				reduceGlobal< 4> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 2:
				reduceGlobal< 2> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			case 1:
				reduceGlobal< 1> << < dimGrid, dimBlock, smemSize, streams[i] >> > (data, data, currentThreads); break;
			}

			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "reduceGlobal launch failed: %s\n", hipGetErrorString(cudaStatus));
				return cudaStatus;
			}
		}

		currentThreads /= THREADS_PER_BLOCK;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching countMembers!\n", cudaStatus);
		return cudaStatus;
	}

	int blocks = (int)ceil((float)k / THREADS_PER_BLOCK);
	gather << <blocks, THREADS_PER_BLOCK >> > (values, n, k);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "reduceGlobal launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching countMembers!\n", cudaStatus);
		return cudaStatus;
	}

	return hipSuccess;
}

void coalesceData(float** points, int pointsCount, int dimNum) {

	float* coalesced = new float[pointsCount * dimNum];
	int64_t indx = 0;

	for (size_t i = 0; i < dimNum; i++)
	{
		for (size_t j = 0; j < pointsCount; j++)
		{
			coalesced[indx++] = (*points)[j * dimNum + i];
		}
	}

	free(*points);
	*points = coalesced;
}

void unCoalesceData(float** points, int pointsCount, int dimNum) {

	float* unCoalesced = new float[pointsCount * dimNum];

	for (size_t i = 0; i < pointsCount; i++)
	{
		for (size_t j = 0; j < dimNum; j++)
		{
			unCoalesced[i * dimNum + j] = (*points)[j * pointsCount + i];
		}
	}

	free(*points);
	*points = unCoalesced;
}

void generateClusters(float* minCoordinates, float* maxCoordinates, float** clusters, int clusterCount, int dimNum) {

	*clusters = new float[clusterCount * dimNum];


	for (size_t i = 0; i < dimNum; i++)
	{
		float diff = maxCoordinates[i] - minCoordinates[i];

		for (size_t j = 0; j < clusterCount; j++)
		{
			float random = (float)rand() / (float)RAND_MAX;
			(*clusters)[i * clusterCount + j] = random * diff + minCoordinates[i];
		}
	}

}

void getMinMax(float* points, int pointsCount, int dimNum, float** maxCoordinates, float** minCoordinates) {

	(*minCoordinates) = (float*)malloc(dimNum * sizeof(float));
	(*maxCoordinates) = (float*)malloc(dimNum * sizeof(float));
	memset(*minCoordinates, FLT_MAX, dimNum * sizeof(float));
	memset(*maxCoordinates, FLT_MIN, dimNum * sizeof(float));


	for (size_t i = 0; i < dimNum; i++)
	{
		for (size_t j = 0; j < pointsCount; j++)
		{
			auto coordinate = points[i * pointsCount + j];

			if ((*maxCoordinates)[i] < coordinate)
				(*maxCoordinates)[i] = coordinate;
			if ((*minCoordinates)[i] > coordinate)
				(*minCoordinates)[i] = coordinate;
		}
	}
}

float* readFile(
	char* filename,
	int* pointsCount,
	int* dimNum
)
{
	float* points;
	int     i, j, len;
	FILE* infile;
	char* line, * ret;
	int   lineLen;

	if ((infile = fopen(filename, "r")) == NULL) {
		fprintf(stderr, "Error: no such file (%s)\n", filename);
		return NULL;
	}

	/* first find the number of objects */
	lineLen = MAX_CHAR_PER_LINE;
	line = (char*)malloc(lineLen);
	assert(line != NULL);

	(*pointsCount) = 0;
	while (fgets(line, lineLen, infile) != NULL) {
		/* check each line to find the max line length */
		while (strlen(line) == lineLen - 1) {
			/* this line read is not complete */
			len = (int)strlen(line);
			fseek(infile, -len, SEEK_CUR);

			/* increase lineLen */
			lineLen += MAX_CHAR_PER_LINE;
			line = (char*)realloc(line, lineLen);
			assert(line != NULL);

			ret = fgets(line, lineLen, infile);
			assert(ret != NULL);
		}

		if (strtok(line, " \t\n") != 0)
			(*pointsCount)++;
	}
	rewind(infile);

	/* find the no. objects of each object */
	(*dimNum) = 0;
	while (fgets(line, lineLen, infile) != NULL) {
		if (strtok(line, " \t\n") != 0) {
			/* ignore the id (first coordiinate): numCoords = 1; */
			while (strtok(NULL, " ,\t\n") != NULL) (*dimNum)++;
			break; /* this makes read from 1st object */
		}
	}
	rewind(infile);
	if (DEBUG) {
		printf("Points = %d\n", *pointsCount);
		printf("Dimensions = %d\n", *dimNum);
	}

	/* allocate space for objects[][] and read all objects */
	len = (*pointsCount) * (*dimNum);
	points = (float*)malloc((*pointsCount) * (*dimNum) * sizeof(float*));


	i = 0;
	/* read all objects */
	while (fgets(line, lineLen, infile) != NULL) {
		if (strtok(line, " \t\n") == NULL) continue;
		for (j = 0; j < (*dimNum); j++) {
			float coordinate = (float)atof(strtok(NULL, " ,\t\n"));

			points[i * (*dimNum) + j] = coordinate;
		}
		i++;
	}

	fclose(infile);
	free(line);

	return points;
}


void printClusters(float* points, int pointsCount, int dimNum) {

	for (size_t i = 0; i < pointsCount; i++)
	{
		printf("%d. ", i + 1);

		for (size_t j = 0; j < dimNum; j++)
		{
			printf("%f ", points[i * dimNum + j]);
		}

		printf("\n");
	}
}

int writeFile(char* filename,
	int clustersCount,
	int pointsCount,
	int dimNum,
	float* clusters,
	int* membership)
{
	FILE* fptr;
	int   i, j;
	char  outFileName[1024];

	sprintf(outFileName, "%s.cluster_centres", filename);
	printf("Writing coordinates of K=%d cluster centers to file \"%s\"\n",
		clustersCount, outFileName);
	fptr = fopen(outFileName, "w");
	for (i = 0; i < clustersCount; i++) {
		fprintf(fptr, "%d ", i);
		for (j = 0; j < dimNum; j++)
			fprintf(fptr, "%f ", clusters[i * dimNum + j]);
		fprintf(fptr, "\n");
	}
	fclose(fptr);

	sprintf(outFileName, "%s.membership", filename);
	printf("Writing membership of N=%d data objects to file \"%s\"\n",
		pointsCount, outFileName);
	fptr = fopen(outFileName, "w");
	for (i = 0; i < pointsCount; i++)
		fprintf(fptr, "%d %d\n", i, membership[i]);
	fclose(fptr);

	return 1;
}

void generateRandomPoints(char* path, int pointsCount, int dimNum) {

	FILE* outFile = NULL;

	if ((outFile = fopen(path, "w")) == NULL) {
		fprintf(stderr, "Error: Cannot create a new file (%s)\n", path);
		return;
	}

	float min = -5;
	float max = 5;
	float diff = max - min;

	for (size_t i = 0; i < pointsCount; i++)
	{
		fprintf(outFile, "%d ", i);

		for (size_t j = 0; j < dimNum; j++)
		{
			float random = (float)rand() / (float)RAND_MAX;
			fprintf(outFile, "%f ", random * diff + min);
		}

		fprintf(outFile, "\n");
	}
}

void generateRandomPointsNormal(char* path, int pointsCount, int dimNum, int groups) {

	FILE* outFile = NULL;

	if ((outFile = fopen(path, "w")) == NULL) {
		fprintf(stderr, "Error: Cannot create a new file (%s)\n", path);
		return;
	}

	float min = -5;
	float max = 5;
	float diff = max - min;

	std::random_device rd{};
	std::mt19937 gen{ rd() };

	//for (size_t i = 0; i < pointsCount; i++)
	//{
	//	fprintf(outFile, "%d ", i);

	//	for (size_t j = 0; j < dimNum; j++)
	//	{
	//		float random = (float)rand() / (float)RAND_MAX;
	//		fprintf(outFile, "%f ", random * diff + min);
	//	}

	//	fprintf(outFile, "\n");
	//}

	for (size_t g = 0; g < groups; g++)
	{
		std::normal_distribution<> d{ -(float)groups / 2 + g, 0.2 };

		for (size_t i = 0; i < pointsCount / groups; i++)
		{
			fprintf(outFile, "%d ", i);

			for (size_t j = 0; j < dimNum; j++)
			{
				fprintf(outFile, "%f ", d(gen));
			}

			fprintf(outFile, "\n");
		}
	}

}